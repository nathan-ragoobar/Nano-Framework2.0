#include <unistd.h>
#include <iostream>
#include <memory>

#include <nvtx3/nvToolsExt.h>
#include "./gpt2.hpp"
#include "llmc/dataloader.h"
#include "llmc/tokenizer.h"
//#include "optim.hpp"
#include  "./nano.hpp"
#include "cuda_profile_util.hpp"
#include <eigen/unsupported/Eigen/CXX11/Tensor>
#include <hip/hip_runtime.h>


// sampler

unsigned int random_u32(unsigned long long* state) {
  // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
  *state ^= *state >> 12;
  *state ^= *state << 25;
  *state ^= *state >> 27;
  return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32(unsigned long long* state) {  // random float32 in [0,1)
  return (random_u32(state) >> 8) / 16777216.0f;
}

int sample_mult(float* probabilities, int n, float coin) {
  // sample index from probabilities (they must sum to 1!)
  // coin is a random number in [0, 1), usually from random_f32()
  float cdf = 0.0f;
  for (int i = 0; i < n; i++) {
    cdf += probabilities[i];
    if (coin < cdf) {
      return i;
    }
  }
  return n - 1;  // in case of rounding errors
}

// CUDA error checking
void cudaCheck(hipError_t error, const char* file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

int main(int argc, char** argv) {

  gpt2::GPT2Config config;
  config.max_seq_len = 1024;
  config.vocab_size = 50257;
  config.padded_vocab_size = 50304;
  config.num_layers = 12;
  config.num_heads = 12;
  config.channels = 768;

  gpt2::GPT2 model;
  //model.InitializeFromScratch(config);
  model.BuildFromCheckpoint("gpt2_124M.bin");

  // build the DataLoaders from tokens files. for now use tiny_shakespeare if
  // available, else tiny_stories
  const char* tiny_stories_train = "dev/data/tinystories/TinyStories_train.bin";
  const char* tiny_stories_val = "dev/data/tinystories/TinyStories_val.bin";
  const char* tiny_shakespeare_train =
      "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
  const char* tiny_shakespeare_val =
      "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
  const char* train_tokens = access(tiny_stories_train, F_OK) != -1
      ? tiny_stories_train
      : tiny_shakespeare_train;
  const char* val_tokens = access(tiny_stories_val, F_OK) != -1
      ? tiny_stories_val
      : tiny_shakespeare_val;
  int B = 4;   // batch size 4 (i.e. 4 independent token sequences will be
               // trained on)
  int T = 64;  // sequence length 64 (i.e. each sequence is 64 tokens long).
               // must be <= maxT, which is 1024 for GPT-2
  DataLoader train_loader, val_loader;
  dataloader_init(&train_loader, train_tokens, B, T, 0, 1, 0);
  dataloader_init(&val_loader, val_tokens, B, T, 0, 1, 0);
  printf("train dataset num_batches: %zu\n", train_loader.num_tokens / (B * T));
  printf("val dataset num_batches: %zu\n", val_loader.num_tokens / (B * T));
  int val_num_batches = 5;

  // build the Tokenizer
  Tokenizer tokenizer;
  tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

  // some memory for generating samples from the model
  unsigned long long rng_state = 1337;
  int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
  const int genT = 64;  // number of steps of inference we will do

  // train
  struct timespec start, end;
  int V = model.config.vocab_size;
  std::unique_ptr<float[]> prob = std::make_unique<float[]>(B * T * V);
  std::unique_ptr<float[]> label = std::make_unique<float[]>(B * T * V);

  // After Parameter creation
    //printf("Device memory info before allocation:\n");
    //size_t free_mem, total_mem;
    //cudaCheck(hipMemGetInfo(&free_mem, &total_mem));
    //printf("Free: %zu MB, Total: %zu MB\n", free_mem/(1024*1024), total_mem/(1024*1024));



  nn::Parameter d_label(nn::DT_FLOAT, B * T * V),
      d_logit(nn::DT_FLOAT, B * T * V), d_prob(nn::DT_FLOAT, B * T * V);
/*
    // Verify each Parameter's memory
    printf("d_label ptr: %p, size: %zu bytes\n", 
        (void*)d_label.data<float>(), 
        d_label.size() * sizeof(float));
    printf("d_logit ptr: %p, size: %zu bytes\n",
        (void*)d_logit.data<float>(),
        d_logit.size() * sizeof(float));
    printf("d_prob ptr: %p, size: %zu bytes\n",
        (void*)d_prob.data<float>(),
        d_prob.size() * sizeof(float));
*/

  // Add after Parameter creation
  //Check if memory allocation is successful
   if (!d_label.data<float>() || !d_logit.data<float>() || !d_prob.data<float>()) {
    printf("GPU memory allocation failed\n");
    exit(1);
}    


  nn::Softmax softmax;
  std::vector<nn::Parameter*> parameters;
  model.Parameters(&parameters);
  optim::AdamW optimizer(parameters, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.0f);
  std::vector<double> timings;
  for (int step = 0; step <= 40; step++) {
    NvtxRange step_range("Train step", step);

    // once in a while estimate the validation loss
    if (step % 10 == 0) {
      float val_loss = 0.0f;
      dataloader_reset(&val_loader);
      for (int i = 0; i < val_num_batches; i++) {
        NvtxRange validation_range("validation");
        dataloader_next_batch(&val_loader);
        float loss = 0.0f;
        auto idx = TTypes<int>::ConstMatrix(val_loader.inputs, B, T);
        std::memset(label.get(), 0, sizeof(float) * B * T * V);
        nn::OntHot(MakeConstFlat(val_loader.targets, B * T),
                   MakeMatrix(label.get(), B * T, V));


                   

        // Before hipMemcpy, add synchronization and validation
        cudaCheck(hipDeviceSynchronize());
        if (!label.get()) {
            printf("Host memory is null\n");
            exit(1);
        }
        /*
        printf("Debug: Copying %zu bytes from host(%p) to device(%p)\n", 
            sizeof(float) * B * T * V, 
            (void*)label.get(), 
            (void*)d_label.data<float>());
          */
        // Before the hipMemcpy, verify alignment
        if ((reinterpret_cast<std::uintptr_t>(d_label.data<float>()) % 16) != 0) {
            printf("Warning: Device pointer not 16-byte aligned\n");
        }
        if ((reinterpret_cast<std::uintptr_t>(label.get()) % 16) != 0) {
            printf("Warning: Host pointer not 16-byte aligned\n");
}


    // Before the problematic hipMemcpy
    //cudaCheck(hipDeviceReset());  // Reset device state
    //cudaCheck(hipSetDevice(0));   // Ensure we're on the right device

    // Add error checking before memcpy
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error before memcpy: %s\n", hipGetErrorString(err));
    }

    // Try pinned memory for the host
    float* pinned_label;
    cudaCheck(hipHostMalloc(&pinned_label, sizeof(float) * B * T * V));
    std::memcpy(pinned_label, label.get(), sizeof(float) * B * T * V);

    // Copy from pinned memory to device (correct direction)
    cudaCheck(hipMemcpy(d_label.data<float>(),  // destination (device)
    pinned_label,            // source (host)
    sizeof(float) * B * T * V,
    hipMemcpyHostToDevice));

    // Free pinned memory
    cudaCheck(hipHostFree(pinned_label));

/*
        cudaCheck(hipMemcpy(d_label.data<float>(), label.get(),
                            sizeof(float) * B * T * V,
                            hipMemcpyHostToDevice));
        cudaCheck(hipDeviceSynchronize());
*/


            /*
        cudaCheck(hipMemcpy(d_label.data<float>(), label.get(),
                             sizeof(float) * B * T * V,
                             hipMemcpyHostToDevice));
                             */
        auto label_3d = d_label.const_tensor_3d<float>(B, T, V);
        auto logit_3d = d_logit.tensor_3d<float>(B, T, V);
        model.gpt2_->ForwardGPU(idx, label_3d, logit_3d, &loss);
        val_loss += loss;
      }
      val_loss /= val_num_batches;

      if (step == 0) {
        size_t num_activations = model.gpt2_->NumActivations();
        printf("num_activations: %zu(%zu MB)\n", num_activations,
               num_activations * sizeof(floatX) / 1024 / 1024);
      }
      printf("val loss %f\n", val_loss);
    }

    // once in a while do model inference to print generated text
    if (step > 0 && step % 20 == 0) {
      NvtxRange generation_range("generation");
      // fill up gen_tokens with the GPT2_EOT, which kicks off the generation
      for (int i = 0; i < B * T; ++i) {
        gen_tokens[i] = tokenizer.eot_token;
      }
      // now sample from the model autoregressively
      printf("generating:\n---\n");
      for (int t = 1; t < genT; t++) {
        // note that inference is very wasteful here because for each token
        // we re-calculate the forward pass for all of (B,T) positions from
        // scratch but the inference here is just for sanity checking anyway and
        // we can maybe optimize a bit more later, with careful tests
        auto gen_tokens_2d = TTypes<int>::ConstMatrix(gen_tokens, B, T);
        auto logit_3d = d_logit.tensor_3d<float>(B, T, V);
        model.gpt2_->Forward(gen_tokens_2d, logit_3d);
        auto logit_2d = d_logit.const_matrix<float>(B * T, V);
        auto prob_2d = d_prob.matrix<float>(B * T, V);
        softmax.Forward(logit_2d, prob_2d);
        nn::g_device.memcpyDeviceToHost(prob.get(), d_prob.data<float>(),
                                        sizeof(float) * B * T * V);
        //nn::g_device.synchronize();
        hipDeviceSynchronize(); //From the CUDA runtime API
        // furthermore, below we're only using b=0 (i.e. the first row) of all B
        // rows we're in principle running B "inference streams" in parallel
        // here but only using position 0 get the Vp-dimensional vector probs[0,
        // t-1, :]
        float* probs = prob.get() + (t - 1) * V;
        float coin = random_f32(&rng_state);
        // note we're only sampling from the first V elements, ignoring padding
        // (the probabilities in the padded region should be zero anyway)
        int next_token = sample_mult(probs, model.config.vocab_size, coin);
        gen_tokens[t] = next_token;
        // print the generated token, either using the Tokenizer or a fallback
        if (tokenizer.init_ok) {
          const char* token_str = tokenizer_decode(&tokenizer, next_token);
          safe_printf(token_str);
        } else {
          // fall back to printing the token id
          printf("%d ", next_token);
        }
        fflush(stdout);
      }
      printf("\n---\n");
    }

    // do a training step
    clock_gettime(CLOCK_MONOTONIC, &start);
    dataloader_next_batch(&train_loader);
    float loss = 0.0f;
    auto idx = TTypes<int>::ConstMatrix(train_loader.inputs, B, T);
    std::memset(label.get(), 0, sizeof(float) * B * T * V);
    nn::OntHot(MakeConstFlat(train_loader.targets, B * T),
               MakeMatrix(label.get(), B * T, V));
    cudaCheck(hipMemcpy(d_label.data<float>(), label.get(),
                         sizeof(float) * B * T * V, hipMemcpyHostToDevice));
    auto label_3d = d_label.const_tensor_3d<float>(B, T, V);
    auto logit_3d = d_logit.tensor_3d<float>(B, T, V);
    model.gpt2_->ForwardGPU(idx, label_3d, logit_3d, &loss);
    optimizer.ZeroGrad();
    model.gpt2_->BackwardGPU(idx);
    optimizer.Step(step + 1);
    clock_gettime(CLOCK_MONOTONIC, &end);
    double time_elapsed_s =
        (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

    printf("step %d: train loss %f (took %f ms)\n", step, loss,
           time_elapsed_s * 1000);
    if (step) {
      timings.push_back(time_elapsed_s);
    }
  }

  double sum = std::accumulate(timings.begin(), timings.end(), 0.0);
  if (!timings.empty()) {
    printf("final %zu iters avg: %.3f ms\n", timings.size(),
           1000 * sum / timings.size());
  }

  //Save model
  model.SaveModel("gpt2_124M100Steps.bin");

  // free
  dataloader_free(&train_loader);
  dataloader_free(&val_loader);
  tokenizer_free(&tokenizer);
  free(gen_tokens);
  return 0;
}
